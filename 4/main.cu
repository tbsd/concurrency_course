﻿#include <cmath>
#include <functional>
#include <iomanip>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

double f(double x, double y, double z) {
  return std::cos(3 * x) - 4 * y;
}
double g(double x, double y, double z) {
  return (z);
}

__global__ void calculateK1(float* k1,
                            const float* a,
                            const float* b,
                            float xn,
                            float yn,
                            float h) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  k1[i] = h * (std::cos(a[i] * xn) - b[i] * yn);  // z' = cos(ax) - by
}

__global__ void calculateK2(float* k2,
                            const float* a,
                            const float* b,
                            const float* k1,
                            float xn,
                            float yn,
                            float h) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  k2[i] = h * (std::cos(a[i] * (xn + h / 2)) - b[i] * (yn + k1[i] * h / 2));
}

__global__ void calculateK3(float* k3,
                            const float* a,
                            const float* b,
                            const float* k2,
                            float xn,
                            float yn,
                            float h) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  k3[i] = h * (std::cos(a[i] * (xn + h / 2)) - b[i] * (yn + k2[i] * h / 2));
}

__global__ void calculateK4(float* k4,
                            const float* a,
                            const float* b,
                            const float* k3,
                            float xn,
                            float yn,
                            float h) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  k4[i] = h * (std::cos(a[i] * (xn + h)) - b[i] * (yn + k3[i] * h));
}
__global__ void calculateYn(float* yn,
                            const float* a,
                            const float* b,
                            const float* k1,
                            const float* k2,
                            const float* k3,
                            const float* k4,
                            float h) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  yn[i] = yn[i] + (k1[i] + 2 * k2[i] + 2 * k3[i] + k4[i]) / 6;
}

// Решение дифуров методом Рунге-Кутта
#define BLOCK_SIZE 256
// nvcc --ptxas-options=-v  main.cu -o prog
int main() {
  float x0 = 0;
  float xn = x0;
  std::vector<float> yn = {0.8, 2};
  float h = 0.001;
  int totalSteps = 10000;
  const int system_size = 100;
  std::time_t seed = std::time(nullptr);
  srand(seed);
  std::cout << "Seed: " << seed << std::endl;

  float* d_k1;
  float* d_k2;
  float* d_k3;
  float* d_k4;
  float* d_yn;
  float* d_a;
  float* d_b;
  int N = 100;  // BLOCK_SIZE * 100;
  size_t size = N * sizeof(float);
  float* h_k1 = (float*)malloc(size);
  float* h_k2 = (float*)malloc(size);
  float* h_k3 = (float*)malloc(size);
  float* h_k4 = (float*)malloc(size);
  float* h_yn = (float*)malloc(size);
  float* h_a = (float*)malloc(size);
  float* h_b = (float*)malloc(size);
  for (int i = 0; i < N; ++i) {
    h_a[i] = rand() % 1000;
    h_b[i] = rand() % 1000;
  }

  int threadsPerBlock = BLOCK_SIZE;
  int blocksPerGrid = N / threadsPerBlock;
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
  for (int i = 0; i < totalSteps; ++i) {
    hipMalloc((void**)&d_k1, size);
    calculateK1<<<blocksPerGrid, threadsPerBlock>>>(d_k1, d_a, d_b, xn, yn[0],
                                                    h);
    hipMemcpy(h_k1, d_k1, size, hipMemcpyDeviceToHost);

    hipMemcpy(d_k1, h_k1, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_k2, size);
    calculateK2<<<blocksPerGrid, threadsPerBlock>>>(d_k2, d_a, d_b, d_k1, xn,
                                                    yn[0], h);
    hipMemcpy(h_k2, d_k2, size, hipMemcpyDeviceToHost);

    hipMemcpy(d_k2, h_k2, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_k3, size);
    calculateK3<<<blocksPerGrid, threadsPerBlock>>>(d_k3, d_a, d_b, d_k2, xn,
                                                    yn[0], h);
    hipMemcpy(d_k3, h_k3, size, hipMemcpyDeviceToHost);

    hipMemcpy(d_k3, h_k3, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_k4, size);
    calculateK4<<<blocksPerGrid, threadsPerBlock>>>(d_k4, d_a, d_b, d_k3, xn,
                                                    yn[0], h);
    hipMemcpy(d_k4, h_k4, size, hipMemcpyDeviceToHost);

    hipMemcpy(d_k4, h_k4, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_yn, size);
    calculateYn<<<blocksPerGrid, threadsPerBlock>>>(d_yn, d_a, d_b, d_k1, d_k2,
                                                    d_k3, d_k4, h);
    hipMemcpy(h_yn, d_yn, size, hipMemcpyDeviceToHost);

    xn += h;

    for (int j = 0; j < N; j++)
      std::cout << h_yn[j] << " ";
    std::cout << std::endl;
  }
  //  DiffEquation de(system, x0, y0i, h);
  //  std::vector<Point> results(totalSteps, Point{});
  //  for (int i = 0; i < totalSteps; ++i) {
  //    results[i] = de.next();
  //  }
  //  for (auto& r : results) {
  //    std::cout << r.x << " ";
  //    for (auto y : r.y) {
  //      std::cout << y << " ";
  //    }
  //    std::cout << "\n";
  //  }
  free(h_k1);
  free(h_k2);
  free(h_k3);
  free(h_k4);
  free(h_yn);
  free(h_a);
  free(h_b);
  return 0;
}
